#include "hip/hip_runtime.h"
// Dot Product - Workshop 5
// w5.shared.cu

#include <iostream>
#include <cstdlib>
#include <ctime>
// CUDA header file
#include <hip/hip_runtime.h>
#include "cuPrintf.cu"
using namespace std;


#define imin(a, b) (a < b ? a : b)

void init(float*, int);

const int ntpb = 256; // number of threads per block

// CUDA kernel code
__global__ void dotProduct (float* da, float* db, float* dc, int n) {
  // cuPrintf("__global__ dotProduct\n");
  __shared__ float s_results[ntpb];
  int tid = threadIdx.x;
  int i = blockIdx.x * blockDim.x + tid;
  float temp = 0;
  if (i < n) {
    temp = da[i] * db[i];
  }
  s_results[tid] = temp;

  __syncthreads();
  for (int stride = 1; stride < blockDim.x; stride *= 2) {
    // cuPrintf("########## stride %d ##########\n", stride);
    if (tid % (2 * stride) == 0 && tid + stride < blockDim.x) {
      s_results[tid] += s_results[tid + stride];
      // cuPrintf("s_results[%d] += s_results[%d] = %f\n", tid, tid + stride, s_results[tid + stride]);
     }
    __syncthreads();
  }

  if (tid == 0) {
    // cuPrintf("s_results[0]: %f\n", s_results[0]);
    dc[blockIdx.x] = s_results[0];
    // cuPrintf("dc[%d]: %f\n", blockIdx.x, dc[blockIdx.x]);
  }
}

int main(int argc, char** argv) {
  // interpret command-line arguments
  if (argc != 2) {
    cerr << "**invalid number of arguments**" << endl;
    return 1;
  }
  int n = atoi(argv[1]);
  srand((unsigned)time(NULL));
  int nbpg = imin(32, (n + ntpb - 1) / ntpb);

  // host vectors
  float* ha = new float[n];
  float* hb = new float[n];
  float* hc = new float[1];
  init(ha, n);
  init(hb, n);

  // device vectors (da[n], db[n], dc[n])
  float* da;
  float* db;
  float* dc;

  hipMalloc((void**)&da, n * sizeof(float));
  hipMalloc((void**)&db, n * sizeof(float));
  hipMalloc((void**)&dc, n * sizeof(float));

  // copy from the host to the device ha -> da, hb -> db
  hipMemcpy(da, ha, n * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(db, hb, n * sizeof(float), hipMemcpyHostToDevice);

  cudaPrintfInit();

  dim3 block(ntpb,1,1);
  dim3 grid(nbpg,1,1);

  cout << "number of blocks per grid: " << nbpg << endl;
  cout << "number of threads per block: " << ntpb << endl;

  // calculate the dot product on the device
  dotProduct<<<grid, block>>>(da, db, dc, n);

  cudaPrintfDisplay(stdout, true);
  cudaPrintfEnd();

  // copy the result from the device to the host dc -> hc
  hipMemcpy(hc, dc, n * sizeof(float), hipMemcpyDeviceToHost);

  float dx = 0;
  for (int i = 0; i < nbpg; i++) {
    dx += hc[i];
  }
  // dot product on the host
  float hx = 0;
  for (int i = 0; i < n; i++) {
    hx += ha[i] * hb[i];
  }

  // compare results
  cout << "Device = " << dx << " Host = " << hx << endl;

  // free device memory
  hipFree(da);
  hipFree(db);
  hipFree(dc);


  // free host memory
  delete [] ha;
  delete [] hb;
  delete [] hc;

  return 0;
}

void init(float* a, int n) {
  float f = 1.0f / RAND_MAX;
  for (int i = 0; i < n; i++) {
    a[i] = ::rand() * f; // [0.0f 1.0f]
  }
}