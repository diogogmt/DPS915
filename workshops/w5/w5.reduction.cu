#include "hip/hip_runtime.h"
// Dot Product - Workshop 5
// w5.reduction.cu

#include <iostream>
#include <cstdlib>
#include <ctime>
// CUDA header file
#include <hip/hip_runtime.h>
#include "cuPrintf.cu"
using namespace std;

void init(float*, int);

// CUDA kernel code
__global__ void dotProduct (float* da, float* db, float* dc) {
  // cuPrintf("__global__ dotProduct\n");
  int tid = threadIdx.x;
  dc[tid] = da[tid] * db[tid];

  __syncthreads();
  for (int stride = 1; stride < blockDim.x; stride *= 2) {
    // cuPrintf("########## stride %d ##########\n", stride);
    if (tid % (2 * stride) == 0 && tid + stride < blockDim.x) {
      dc[tid] += dc[tid + stride];
      // cuPrintf("dc[%d] += dc[%d] = %f\n", tid, tid + stride, dc[tid + stride]);
     }
    __syncthreads();
  }
}

int main(int argc, char** argv) {
  // interpret command-line arguments
  if (argc != 2) {
    cerr << "**invalid number of arguments**" << endl;
    return 1;
  }
  int n = atoi(argv[1]);
  srand((unsigned)time(NULL));

  // host vectors
  float* ha = new float[n];
  float* hb = new float[n];
  float* hc = new float[1];
  init(ha, n);
  init(hb, n);

  // device vectors (da[n], db[n], dc[n])
  float* da;
  float* db;
  float* dc;

  hipMalloc((void**)&da, n * sizeof(float));
  hipMalloc((void**)&db, n * sizeof(float));
  hipMalloc((void**)&dc, n * sizeof(float));

  // copy from the host to the device ha -> da, hb -> db
  hipMemcpy(da, ha, n * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(db, hb, n * sizeof(float), hipMemcpyHostToDevice);

  cudaPrintfInit();

  // calculate the dot product on the device
  dotProduct<<<1, n>>>(da, db, dc);

  cudaPrintfDisplay(stdout, true);
  cudaPrintfEnd();

  // copy the result from the device to the host dc -> hc
  hipMemcpy(hc, dc, n * sizeof(float), hipMemcpyDeviceToHost);

  float dx = hc[0];
  // dot product on the host
  float hx = 0;
  for (int i = 0; i < n; i++) {
    hx += ha[i] * hb[i];
  }

  // compare the results
  cout << "Device = " << dx << " Host = " << hx << endl;

  // free device memory

  hipFree(da);
  hipFree(db);
  hipFree(dc);

  // free host memory
  delete [] ha;
  delete [] hb;
  delete [] hc;


  return 0;
}

void init(float* a, int n) {
  float f = 1.0f / RAND_MAX;
  for (int i = 0; i < n; i++) {
    a[i] = ::rand() * f; // [0.0f 1.0f]
    // a[i] = 2; // [0.0f 1.0f]

  }
}