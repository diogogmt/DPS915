#include "hip/hip_runtime.h"
// Dot Product - Workshop 5
// w5.shared.cu

#include <iostream>
#include <cstdlib>
#include <ctime>
// CUDA header file
#include <hip/hip_runtime.h>
#include "cuPrintf.cu"
using namespace std;

void init(float*, int);

const int ntpb = 256; // number of threads per block

// CUDA kernel code
__global__ void dotProduct (float* da, float* db, float* dc, int n) {
  // cuPrintf("__global__ dotProduct\n");
  __shared__ float s_results[ntpb];
  int tid = threadIdx.x;
  int i =  tid;
  float temp = 0;
  if (i < n) {
    temp = da[i] * db[i];
  }
  s_results[tid] = temp;

  __syncthreads();
  for (int stride = 1; stride < blockDim.x; stride *= 2) {
    // cuPrintf("########## stride %d ##########\n", stride);
    if (tid % (2 * stride) == 0 && tid + stride < blockDim.x) {
      s_results[tid] += s_results[tid + stride];
      // cuPrintf("s_results[%d] += s_results[%d] = %f\n", tid, tid + stride, s_results[tid + stride]);
     }
    __syncthreads();
  }

  // Only works because we have only one block of threads
  // If we had more than one block, s_results would be allocated for each block separately
  dc[0] = s_results[0];
}

int main(int argc, char** argv) {
  // interpret command-line arguments
  if (argc != 2) {
    cerr << "**invalid number of arguments**" << endl;
    return 1;
  }
  int n = atoi(argv[1]);
  srand((unsigned)time(NULL));

  // host vectors
  float* ha = new float[n];
  float* hb = new float[n];
  float* hc = new float[1];
  init(ha, n);
  init(hb, n);

  // device vectors (da[n], db[n], dc[n])
  float* da;
  float* db;
  float* dc;

  hipMalloc((void**)&da, n * sizeof(float));
  hipMalloc((void**)&db, n * sizeof(float));
  hipMalloc((void**)&dc, n * sizeof(float));

  // copy from the host to the device ha -> da, hb -> db
  hipMemcpy(da, ha, n * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(db, hb, n * sizeof(float), hipMemcpyHostToDevice);

  cudaPrintfInit();

  // calculate the dot product on the device
  dotProduct<<<1, ntpb>>>(da, db, dc, n);


  cudaPrintfDisplay(stdout, true);
  cudaPrintfEnd();
  // copy the result from the device to the host dc -> hc
  hipMemcpy(hc, dc, n * sizeof(float), hipMemcpyDeviceToHost);

  float hx = 0;
  for (int i = 0; i < n; i++) {
    hx += ha[i] * hb[i];
  }

  // compare results
  cout << "Device = " << hc[0] << " Host = " << hx << endl;

  // free device memory
  hipFree(da);
  hipFree(db);
  hipFree(dc);

  // free host memory
  delete [] ha;
  delete [] hb;
  delete [] hc;

  return 0;
}

void init(float* a, int n) {
  float f = 1.0f / RAND_MAX;
  for (int i = 0; i < n; i++) {
    a[i] = ::rand() * f; // [0.0f 1.0f]
  }
}